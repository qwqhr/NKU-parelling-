#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <chrono>
#define ll long long

const int N = 300050;
const int G = 3;
const int BLOCK_SIZE = 512;
const int MAX_TESTS = 5; 

static ll* d_roots_cache[MAX_TESTS];
static int max_n_cache[MAX_TESTS];
static bool computed[MAX_TESTS];

#define CUDA_CHECK(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(1); \
    } \
} while(0)

void fRead(int *a, int *b, int *n, int *p, int input_id){
    char filename[100];
    sprintf(filename, "data/%d.in", input_id);
    FILE *fin = fopen(filename, "r");
    
    if (!fin) {
        fprintf(stderr, "Error: Cannot open file %s\n", filename);
        return;
    }
    
    fscanf(fin, "%d %d", n, p);
    for (int i = 0; i < *n; i++) {
        fscanf(fin, "%d", &a[i]);
    }
    for (int i = 0; i < *n; i++) {
        fscanf(fin, "%d", &b[i]);
    }
    fclose(fin);
}

void fCheck(int *ab, int n, int input_id){
    char filename[100];
    sprintf(filename, "data/%d.out", input_id);
    FILE *fin = fopen(filename, "r");
    
    if (!fin) {
        fprintf(stderr, "Error: Cannot open file %s\n", filename);
        printf("无法验证结果\n");
        return;
    }
    
    bool correct = true;
    for (int i = 0; i < n * 2 - 1; i++) {
        int expected;
        fscanf(fin, "%d", &expected);
        if (ab[i] != expected) {
            correct = false;
            break;
        }
    }
    fclose(fin);
    
    if (correct) {
        printf("多项式乘法结果正确\n");
    } else {
        printf("多项式乘法结果错误\n");
    }
}

ll _pow(ll a, ll p, ll Mod){
    ll ans = 1, mul = a;
    while(p){
        if(p & 1) ans = (ans * mul) % Mod;
        mul = (mul * mul) % Mod;
        p >>= 1;
    }
    return ans;
}

void compute_roots_for_id(int id, int mod, int maxn) {
    if (computed[id] && max_n_cache[id] >= maxn) {
        return;
    }

    if (computed[id]) {
        CUDA_CHECK(hipFree(d_roots_cache[id]));
    }
    
    int total_roots_needed = 0;
    for (int len = 2; len <= maxn; len <<= 1) {
        total_roots_needed += len >> 1;
    }
    
    ll* d_roots;
    CUDA_CHECK(hipMalloc(&d_roots, total_roots_needed * sizeof(ll)));
    
    ll* h_roots = (ll*)malloc(total_roots_needed * sizeof(ll));
    
    int offset = 0;
    

    for (int len = 2; len <= maxn; len <<= 1) {
        ll W = _pow(G, (mod - 1) / len, mod);
        int mid = len >> 1;
        
        ll current_w = 1;
        for (int k = 0; k < mid; k++) {
            h_roots[offset + k] = current_w;
            current_w = (current_w * W) % mod;
        }
        offset += mid;
    }
    
    CUDA_CHECK(hipMemcpy(d_roots, h_roots, total_roots_needed * sizeof(ll), hipMemcpyHostToDevice));
    
    d_roots_cache[id] = d_roots;
    max_n_cache[id] = maxn;
    computed[id] = true;
    
    free(h_roots);
}

void cleanup_all_roots() {
    for (int i = 0; i < MAX_TESTS; i++) {
        if (computed[i]) {
            CUDA_CHECK(hipFree(d_roots_cache[i]));
            computed[i] = false;
        }
    }
}

__global__ void bit_reverse_kernel(int* data, int* idx, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n && i < idx[i]) {
        int temp = data[i];
        data[i] = data[idx[i]];
        data[idx[i]] = temp;
    }
}

__global__ void ntt_butterfly_kernel(int* data, int len, int mod, ll* roots, int roots_offset, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int mid = len >> 1;
    int blocks_per_len = n / len;
    
    if (tid < blocks_per_len * mid) {
        int block_id = tid / mid;
        int in_block_id = tid % mid;
        
        int base = block_id * len;
        int i = base + in_block_id;
        int j = i + mid;
        
        ll w = roots[roots_offset + in_block_id];
        
        int u = data[i];
        int v = (w * data[j]) % mod;
        data[i] = (u + v) % mod;
        data[j] = (u - v + mod) % mod;
    }
}

__global__ void pointwise_mul_kernel(int* a, int* b, int* result, int n, int mod) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        result[i] = ((ll)a[i] * b[i]) % mod;
    }
}

__global__ void final_scale_kernel(int* data, ll inv, int n, int mod) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        data[i] = (data[i] * inv) % mod;
    }
}

void manual_reverse(int* arr, int start, int end) {
    while (start < end) {
        int temp = arr[start];
        arr[start] = arr[end];
        arr[end] = temp;
        start++;
        end--;
    }
}

void cuda_NTT(int* h_data, int* h_idx, int id, int mod, int type, int maxn) {
    // 确保旋转因子已计算
    if (!computed[id] || max_n_cache[id] < maxn) {
        compute_roots_for_id(id, mod, maxn);
    }
    
    int* d_data;
    int* d_idx;
    
    CUDA_CHECK(hipMalloc(&d_data, maxn * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_idx, maxn * sizeof(int)));
    
    CUDA_CHECK(hipMemcpy(d_data, h_data, maxn * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_idx, h_idx, maxn * sizeof(int), hipMemcpyHostToDevice));
    
    // 位逆序
    int grid_size = (maxn + BLOCK_SIZE - 1) / BLOCK_SIZE;
    bit_reverse_kernel<<<grid_size, BLOCK_SIZE>>>(d_data, d_idx, maxn);
    CUDA_CHECK(hipDeviceSynchronize());
    
    // 蝶形运算
    int roots_offset = 0;
    for(int len = 2; len <= maxn; len <<= 1) {
        int mid = len >> 1;
        int total_butterflies = maxn / len * mid;
        int grid = (total_butterflies + BLOCK_SIZE - 1) / BLOCK_SIZE;
        
        ntt_butterfly_kernel<<<grid, BLOCK_SIZE>>>(d_data, len, mod, 
                                                  d_roots_cache[id], 
                                                  roots_offset, maxn);
        CUDA_CHECK(hipDeviceSynchronize());
        
        roots_offset += mid;
    }
    
    if(type == 1) {
        CUDA_CHECK(hipMemcpy(h_data, d_data, maxn * sizeof(int), hipMemcpyDeviceToHost));
        manual_reverse(h_data, 1, maxn - 1);
        CUDA_CHECK(hipMemcpy(d_data, h_data, maxn * sizeof(int), hipMemcpyHostToDevice));
        
        ll inv = _pow(maxn, mod - 2, mod);
        final_scale_kernel<<<grid_size, BLOCK_SIZE>>>(d_data, inv, maxn, mod);
        CUDA_CHECK(hipDeviceSynchronize());
    }
    
    CUDA_CHECK(hipMemcpy(h_data, d_data, maxn * sizeof(int), hipMemcpyDeviceToHost));
    
    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_idx));
}

int idx[N << 2], MAXN, P, nm;

void poly_multiply(int *a, int *b, int *ab, int n, int p, int id) {
    nm = n + n - 1;
    MAXN = 1, P = 0;
    while(MAXN < nm) MAXN <<= 1, P++;

    for(int i = 0; i < MAXN; ++i) {
        idx[i] = (idx[i >> 1] >> 1) | ((i & 1) << (P - 1));
    }
    
    memset(a + n, 0, sizeof(int) * (MAXN - n));
    memset(b + n, 0, sizeof(int) * (MAXN - n));
    
    cuda_NTT(a, idx, id, p, 0, MAXN);
    cuda_NTT(b, idx, id, p, 0, MAXN);
    
    int* d_a, *d_b, *d_ab;
    CUDA_CHECK(hipMalloc(&d_a, MAXN * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_b, MAXN * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_ab, MAXN * sizeof(int)));
    
    CUDA_CHECK(hipMemcpy(d_a, a, MAXN * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b, MAXN * sizeof(int), hipMemcpyHostToDevice));
    
    int grid = (MAXN + BLOCK_SIZE - 1) / BLOCK_SIZE;
    pointwise_mul_kernel<<<grid, BLOCK_SIZE>>>(d_a, d_b, d_ab, MAXN, p);
    CUDA_CHECK(hipDeviceSynchronize());
    
    CUDA_CHECK(hipMemcpy(ab, d_ab, MAXN * sizeof(int), hipMemcpyDeviceToHost));
    
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_ab));
    
    cuda_NTT(ab, idx, id, p, 1, MAXN);
}

int a[N << 2], b[N << 2], ab[N << 2];

int main(int argc, char *argv[])
{
    CUDA_CHECK(hipSetDevice(0));
    
    // 初始化数组
    memset(computed, false, sizeof(computed));
    
    int test_begin = 0;
    int test_end = 3;
    for(int i = test_begin; i <= test_end; ++i){
        long double ans = 0;
        int n_, p_;
        fRead(a, b, &n_, &p_, i);
        memset(ab, 0, sizeof(ab));
        
        auto Start = std::chrono::high_resolution_clock::now();
        poly_multiply(a, b, ab, n_, p_, i);  // 传入测试ID
        auto End = std::chrono::high_resolution_clock::now();
        
        std::chrono::duration<double,std::ratio<1,1000>>elapsed = End - Start;
        ans += elapsed.count();
        fCheck(ab, n_, i);
        printf("average latency for n = %d p = %d : %Lf (ms)\n", n_, p_, ans);
    }
    
    cleanup_all_roots();
    CUDA_CHECK(hipDeviceReset());
    return 0;
}

/*
32
多项式乘法结果正确
average latency for n = 131072 p = 7340033 : 17.328112 (ms)
多项式乘法结果正确
average latency for n = 131072 p = 104857601 : 16.698695 (ms)
多项式乘法结果正确
average latency for n = 131072 p = 469762049 : 16.841276 (ms)

64
多项式乘法结果正确
average latency for n = 131072 p = 7340033 : 16.965059 (ms)
多项式乘法结果正确
average latency for n = 131072 p = 104857601 : 16.783552 (ms)
多项式乘法结果正确
average latency for n = 131072 p = 469762049 : 16.558547 (ms)

128
多项式乘法结果正确
average latency for n = 131072 p = 7340033 : 17.071224 (ms)
多项式乘法结果正确
average latency for n = 131072 p = 104857601 : 15.379213 (ms)
多项式乘法结果正确
average latency for n = 131072 p = 469762049 : 14.337067 (ms)

256
多项式乘法结果正确
average latency for n = 131072 p = 7340033 : 17.965363 (ms)
多项式乘法结果正确
average latency for n = 131072 p = 104857601 : 14.355205 (ms)
多项式乘法结果正确
average latency for n = 131072 p = 469762049 : 15.899327 (ms)

512
多项式乘法结果正确
average latency for n = 131072 p = 7340033 : 17.789513 (ms)
多项式乘法结果正确
average latency for n = 131072 p = 104857601 : 14.523976 (ms)
多项式乘法结果正确
average latency for n = 131072 p = 469762049 : 16.861739 (ms)
*/